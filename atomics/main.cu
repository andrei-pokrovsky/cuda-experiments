#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sstream>

#include <nvToolsExt.h>

#include <unistd.h>

#include "common/common.hpp"

template <typename data_type, size_t REPEATS = 1024>
__global__ void
gpu_touch(data_type *__restrict__ hist, const size_t *__restrict__ idx,
          double *__restrict__ threadTimes, const bool noop = false)
{
  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // where to increment
  const size_t voteIdx = idx[gx];

  hist[voteIdx] = 0;
  __syncthreads();

  const long long int start = clock64();
  if (!noop)
  {
#pragma unroll REPEATS
    for (size_t iter = 0; iter < REPEATS; ++iter)
    {
      atomicAdd(&hist[voteIdx], gx);
    }
  }
  const long long int end = clock64();

  threadTimes[gx] = double(end - start) / REPEATS;
}

int main(void)
{

  const long pageSize = sysconf(_SC_PAGESIZE);
  std::stringstream buffer;

  RT_CHECK(hipFree(0));

  size_t memTotal, memAvail;
  RT_CHECK(hipMemGetInfo(&memAvail, &memTotal));

  typedef int data_type;

  // number of warps making atomic accesses to same location
  // const size_t interWarpConflict;
  // number of threads w/in a warp making atomic accesses
  // const size_t intraWarpConflict;

  for (size_t stride = sizeof(data_type); stride <= 1024 * sizeof(data_type);
       stride *= 2)
  {
    for (size_t numWay = 4; numWay <= 4; ++numWay)
    {

      assert(stride % sizeof(data_type) == 0);

      buffer.str("");
      buffer << "w=" << numWay << " s=" << stride;
      nvtxRangePush(buffer.str().c_str());

      nvtxRangePush("setup");

      // Number of threads
      dim3 dimGrid(1);
      dim3 dimBlock(32);
      const size_t numThreads = dimGrid.x * dimBlock.x;

      // Allocation tracking of thread times
      double *threadTimes_h, *threadTimes_d;
      threadTimes_h = new double[numThreads];
      RT_CHECK(hipMalloc(&threadTimes_d, numThreads * sizeof(double)));

      // Allocate the histogram
      data_type *hist;
      RT_CHECK(hipMalloc(&hist, stride * numThreads * sizeof(data_type)));

      // Allocate thread access indices
      size_t *idx_h, *idx_d;
      idx_h = new size_t[numThreads];
      RT_CHECK(hipMalloc(&idx_d, numThreads * sizeof(size_t)));

      // initialize thread access indices
      for (size_t i = 0; i < numThreads; ++i)
      {
        size_t lx = i % 32;
        size_t idx;
        if (lx < numWay)
        {
          idx = 0;
        }
        else
        {
          idx = i;
        }
        idx_h[i] = idx * (stride / sizeof(data_type));
      }

      // Copy thread access indices to device
      RT_CHECK(hipMemcpy(idx_d, idx_h, numThreads * sizeof(float),
                          hipMemcpyDefault));
      RT_CHECK(hipDeviceSynchronize());
      nvtxRangePop();

      // work loop
      nvtxRangePush("work");
      gpu_touch<<<dimGrid, dimBlock>>>(hist, idx_d, threadTimes_d);
      RT_CHECK(hipGetLastError());
      nvtxRangePop();

      // Get thread times back
      RT_CHECK(hipMemcpy(threadTimes_h, threadTimes_d,
                          sizeof(double) * numThreads, hipMemcpyDefault));

      // Average, min, max thread times
      const double maxCycles =
          *std::max_element(threadTimes_h, &threadTimes_h[numThreads]);

      // std::cout << "s=" << stride << ": "
      //           << "w=" << numWay << ": " << maxCycles << "\n";
      std::cout << stride << ", " << numWay << ", " << maxCycles << "\n";

      // free memory
      nvtxRangePush("cleanup");
      RT_CHECK(hipFree(hist));
      RT_CHECK(hipFree(idx_d));
      RT_CHECK(hipFree(threadTimes_d));
      delete[] idx_h;
      delete[] threadTimes_h;
      nvtxRangePop();

      nvtxRangePop();
    }
  }
  return 0;
}
