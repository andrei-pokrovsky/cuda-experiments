#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sstream>

#include <nvToolsExt.h>

#include <unistd.h>

#include "common/common.hpp"

template <typename data_type, size_t REPEATS = 512>
__global__ void gpu_touch(data_type *hist, const size_t *idx,
                          double *threadTimes, const bool noop = false) {
  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // where to increment
  const size_t voteIdx = idx[gx];

  const long long int start = clock64();
  if (!noop) {
#pragma unroll(REPEATS)
    for (size_t iter = 0; iter < REPEATS; ++iter) {
      atomicAdd(&hist[voteIdx], gx);
    }
  }
  const long long int end = clock64();

  threadTimes[gx] = double(end - start) / REPEATS;
}

int main(void) {

  const long pageSize = sysconf(_SC_PAGESIZE);
  std::stringstream buffer;

  RT_CHECK(hipFree(0));

  size_t memTotal, memAvail;
  RT_CHECK(hipMemGetInfo(&memAvail, &memTotal));

  typedef int data_type;

  const int numIters = 1000000;

  // number of warps making atomic accesses to same location
  // const size_t interWarpConflict;
  // number of threads w/in a warp making atomic accesses
  // const size_t intraWarpConflict;

  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

  float elapsed;

  for (size_t stride = sizeof(data_type); stride <= 64 * sizeof(data_type);
       stride *= 2) {
    for (size_t numWay = 1; numWay <= 32; ++numWay) {

      assert(stride % sizeof(data_type) == 0);

      buffer.str("");
      buffer << "w=" << numWay << " s=" << stride;
      nvtxRangePush(buffer.str().c_str());

      nvtxRangePush("setup");

      // Number of threads
      dim3 dimGrid(1);
      dim3 dimBlock(32);
      const size_t numThreads = dimGrid.x * dimBlock.x;
      const size_t numAtomics = numThreads * numIters;

      // Allocation tracking of thread times
      double *threadTimes_h, *threadTimes_d;
      threadTimes_h = new double[numThreads];
      RT_CHECK(hipMalloc(&threadTimes_d, numThreads * sizeof(double)));

      // Allocate the histogram
      data_type *hist;
      RT_CHECK(hipMalloc(&hist, stride * numThreads * sizeof(data_type)));

      // Allocate thread access indices
      size_t *idx_h, *idx_d;
      idx_h = new size_t[numThreads];
      RT_CHECK(hipMalloc(&idx_d, numThreads * sizeof(size_t)));

      // initialize thread access indices
      for (size_t i = 0; i < numThreads; ++i) {
        size_t lx = i % 32;
        size_t idx;
        if (lx < numWay) {
          idx = 0;
        } else {
          idx = i;
        }
        idx_h[i] = idx * (stride / sizeof(data_type));
      }

      // Copy thread access indices to device
      RT_CHECK(hipMemcpy(idx_d, idx_h, numThreads * sizeof(float),
                          hipMemcpyDefault));
      RT_CHECK(hipDeviceSynchronize());
      nvtxRangePop();

      // work loop
      nvtxRangePush("work");
      gpu_touch<<<dimGrid, dimBlock>>>(hist, idx_d, threadTimes_d);
      RT_CHECK(hipGetLastError());
      nvtxRangePop();

      // Get thread times back
      RT_CHECK(hipMemcpy(threadTimes_h, threadTimes_d,
                          sizeof(double) * numThreads, hipMemcpyDefault));

      // Average, min, max thread times
      double maxCycles =
          *std::max_element(threadTimes_h, &threadTimes_h[numThreads]);

      // std::cout << "s=" << stride << ": "
      //           << "w=" << numWay << ": " << maxCycles << "\n";
      std::cout << stride << ", " << numWay << ", " << maxCycles << "\n";

      // free memory
      nvtxRangePush("cleanup");
      RT_CHECK(hipFree(hist));
      RT_CHECK(hipFree(idx_d));
      RT_CHECK(hipFree(threadTimes_d));
      delete[] idx_h;
      delete[] threadTimes_h;
      nvtxRangePop();

      nvtxRangePop();
    }
  }
  return 0;
}
