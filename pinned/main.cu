#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <numeric>
#include <sstream>
#include <vector>

#include <numa.h>

#include <nvToolsExt.h>

#include <unistd.h>

#include "common/common.hpp"

static void pinned_bw(const Device &dst, const Device &src, const size_t count)
{

  assert((src.is_cpu()) ^ (dst.is_cpu()));

  void *devPtr, *hostPtr;
  void *srcPtr, *dstPtr;

  if (src.is_cpu())
  {
    RT_CHECK(hipSetDevice(dst.id()));
  }
  else
  {
    RT_CHECK(hipSetDevice(src.id()));
  }

  RT_CHECK(hipFree(0));
  RT_CHECK(hipMalloc(&devPtr, count))
  RT_CHECK(hipHostMalloc(&hostPtr, count));

  if (src.is_gpu())
  {
    srcPtr = hostPtr;
    dstPtr = devPtr;
  }
  else
  {
    srcPtr = devPtr;
    dstPtr = hostPtr;
  }

  std::vector<double> times;
  const size_t numIters = 20;
  for (size_t i = 0; i < numIters; ++i)
  {
    nvtxRangePush("dst");
    auto start = std::chrono::high_resolution_clock::now();
    RT_CHECK(hipMemcpy(dstPtr, srcPtr, count, hipMemcpyDefault));
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> txSeconds = end - start;
    nvtxRangePop();
    times.push_back(txSeconds.count());
  }

  const double minTime = *std::min_element(times.begin(), times.end());
  const double avgTime =
      std::accumulate(times.begin(), times.end(), 0.0) / times.size();

  printf(",%.2f", count / 1024.0 / 1024.0 / minTime);
  RT_CHECK(hipHostFree(hostPtr));
  RT_CHECK(hipFree(devPtr));
}

int main(void)
{

  const size_t numNodes = numa_max_node();

  const long pageSize = sysconf(_SC_PAGESIZE);

  std::vector<Device> gpus = get_gpus();
  std::vector<Device> cpus = get_cpus();

  // print header
  printf("Transfer Size (MB)");
  for (const auto cpu : cpus)
  {
    for (const auto gpu : gpus)
    {
      printf(",%s:%s", cpu.name().c_str(), gpu.name().c_str());
    }
  }

  printf("\n");

  for (size_t count = 2048; count <= 1 * 1024ul * 1024ul * 1024ul; count *= 2)
  {
    printf("%f", count / 1024.0 / 1024.0);
    for (const auto cpu : cpus)
    {
      for (const auto gpu : gpus)
      {
        pinned_bw(cpu, gpu, count);
      }
    }

    printf("\n");
  }

  return 0;
}
