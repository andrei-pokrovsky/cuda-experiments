
#include <hip/hip_runtime.h>
#include <cstdio>

int main(void) {
    int n;
    hipError_t err = hipGetDeviceCount(&n);
    if (err != hipSuccess) {
        return -1;
    }

    for (int i = 0; i < n; ++i) {
        printf("%d ",i);
    }
    printf("\n");
}