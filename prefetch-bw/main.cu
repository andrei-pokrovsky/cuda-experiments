#include <cstdio>
#include <cstdlib>
#include <sstream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <numeric>

#include <nvToolsExt.h>

#include <unistd.h>

#include "common/common.hpp"

static void prefetch_bw(const int dstDev, const int srcDev, const size_t count)
{

  if (srcDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(srcDev));
    RT_CHECK(hipFree(0));
  }

  if (dstDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(dstDev));
    RT_CHECK(hipFree(0));
  }

  if (srcDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(srcDev));
  }

  void *ptr;

  RT_CHECK(hipMallocManaged(&ptr, count));

  std::vector<double> times;
  const size_t numIters = 20;
  for (size_t i = 0; i < numIters; ++i)
  {
    // Try to get allocation on source
    nvtxRangePush("move to src");
    RT_CHECK(hipMemPrefetchAsync(ptr, count, srcDev));
    RT_CHECK(hipDeviceSynchronize());
    nvtxRangePop();

    // Prefetch to device and time.
    nvtxRangePush("tx");
    auto start = std::chrono::high_resolution_clock::now();
    RT_CHECK(hipMemPrefetchAsync(ptr, count, dstDev));
    RT_CHECK(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    nvtxRangePop();
    std::chrono::duration<double> txSeconds = end - start;
    times.push_back(txSeconds.count());
  }

  const double minTime = *std::min_element(times.begin(), times.end());
  const double avgTime = std::accumulate(times.begin(), times.end(), 0.0) / times.size();

  printf(",%.2f", count / 1024.0 / 1024.0 / (minTime));

  RT_CHECK(hipFree(ptr));
}

int main(void)
{
  const long pageSize = sysconf(_SC_PAGESIZE);

  int numDevs;
  RT_CHECK(hipGetDeviceCount(&numDevs));

  std::vector<int> devIds;
  for (int dev = 0; dev < numDevs; ++dev)
  {
    devIds.push_back(dev);
  }
  devIds.push_back(hipCpuDeviceId);

  // print header
  printf("Transfer Size (MB),");
  for (const auto src : devIds)
  {
    for (const auto dst : devIds)
    {
      if (src != dst)
      {
        printf("%d:%d,", src, dst);
      }
    }
  }
  printf("\n");

  for (size_t count = 2048; count <= 4 * 1024ul * 1024ul * 1024ul; count *= 2)
  {
    printf("%f", count / 1024.0 / 1024.0);
    for (const auto src : devIds)
    {
      for (const auto dst : devIds)
      {
        if (src != dst)
        {
          prefetch_bw(dst, src, count);
        }
      }
    }
    printf("\n");
  }

  return 0;
}
