#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <sstream>
#include <chrono>
#include <vector>

#include <nvToolsExt.h>

#include <unistd.h>

#include "common/common.hpp"

static void prefetch_bw(const int dstDev, const int srcDev, const size_t count, const size_t stride)
{

  if (srcDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(srcDev));
    RT_CHECK(hipFree(0));
  }

  if (dstDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(dstDev));
    RT_CHECK(hipFree(0));
  }

  if (srcDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(srcDev));
  }

  void *ptr;

  RT_CHECK(hipMallocManaged(&ptr, count));

  double totalWork = 0;
  const size_t numIters = 20;
  for (size_t i = 0; i < numIters; ++i)
  {
    // Try to get allocation on source
    nvtxRangePush("move to src");
    RT_CHECK(hipMemPrefetchAsync(ptr, count, srcDev));
    RT_CHECK(hipDeviceSynchronize());
    nvtxRangePop();

    // Prefetch to device and time.
    nvtxRangePush("tx");
    auto start = std::chrono::high_resolution_clock::now();
    RT_CHECK(hipMemPrefetchAsync(ptr, count, dstDev));
    RT_CHECK(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    nvtxRangePop();
    std::chrono::duration<double> txSeconds = end - start;
    totalWork += txSeconds.count();
  }

  std::cout << "," << count / 1024.0 / 1024.0 / (totalWork / numIters);
  RT_CHECK(hipFree(ptr));
}

int main(void)
{

  const long pageSize = sysconf(_SC_PAGESIZE);

  int numDevs;
  RT_CHECK(hipGetDeviceCount(&numDevs));

  std::vector<int> devIds;
  for (int dev = 0; dev < 3; ++dev)
  {
    devIds.push_back(dev);
  }
  devIds.push_back(hipCpuDeviceId);

  // print header
  std::cout << "Transfer Size (MB),";
  for (const auto src : devIds)
  {
    for (const auto dst : devIds)
    {
      if (src != dst)
      {
	std::cout << src << ":" << dst <<",";
      }
    }
  }
  std::cout << "\n";

  for (size_t count = 2048; count <= 4 * 1024ul * 1024ul * 1024ul; count *= 2) {
    std::cout << count / 1024.0 / 1024.0;
    for (const auto src : devIds)
    {
      for (const auto dst : devIds)
      {
        if (src != dst)
        {
            prefetch_bw(dst, src, count, pageSize);
        }
      }
    }
            std::cout << "\n";
  }

  return 0;
}
