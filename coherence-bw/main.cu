#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <cstdlib>
#include <sstream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <numeric>
#include <cstring>

#include <nvToolsExt.h>

#include <unistd.h>
#include <omp.h>
#include <numa.h>

#include "common/cuda_check.hpp"
#include "common/common.hpp"

static int get_num_cpus(const Device &d)
{
  bitmask *mask = numa_allocate_cpumask();
  numa_node_to_cpus(d.id(), mask);
  int num_cpus = numa_bitmask_weight(mask);
  numa_free_cpumask(mask);
  return num_cpus;
}

template <typename data_type>
void cpu_write(data_type *ptr, const size_t count, const size_t stride)
{

  const size_t numElems = count / sizeof(data_type);
  const size_t elemsPerStride = stride / sizeof(data_type);

#pragma omp parallel for schedule(static)
  for (size_t i = 0; i < numElems; i += elemsPerStride)
  {
    ptr[i] = i * 31ul + 7ul;
  }
}

template <typename data_type, bool NOOP = false>
__global__ void gpu_write(data_type *ptr, const size_t count, const size_t stride)
{
  if (NOOP)
  {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx = gx / 32;
  // number of warps in the grid
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;
  // number of strides in N bytes
  const size_t numStrides = count / stride;
  const size_t numData = count / sizeof(data_type);
  // number of data_types in each
  const size_t dataPerStride = stride / sizeof(data_type);

  if (0 == lx)
  {

    for (; wx < numStrides; wx += numWarps)
    {
      const size_t id = wx * dataPerStride;
      if (id < numData)
      {
        ptr[id] = id * 31ul + 7ul;
      }
    }
  }
}

static void coherence_bw(const Device &dstDev, const Device &srcDev, const size_t count, const size_t stride, const int numIters)
{

  assert(!(srcDev.is_cpu() && dstDev.is_cpu()));

  // Determine grid dimensions
  dim3 blockDim(256);
  const size_t numStrides = (count + stride - 1) / stride;
  dim3 gridDim((numStrides + blockDim.x - 1) / blockDim.x);

  void *ptr;

  // bind to non-cuda device, if present, before allocation or running
  if (srcDev.is_cpu())
  {
    bind_cpu(srcDev);
    omp_set_num_threads(get_num_cpus(srcDev));
#pragma omp parallel
    {
      bind_cpu(srcDev);
    }
  }
  else if (dstDev.is_cpu())
  {
    bind_cpu(dstDev);
    omp_set_num_threads(get_num_cpus(dstDev));
#pragma omp parallel
    {
      bind_cpu(dstDev);
    }
  }

  RT_CHECK(hipMallocManaged(&ptr, count));
  std::memset(ptr, 0, count); // force pages to be allocated

  // If dst is GPU, set that to be the active device before running
  if (dstDev.is_gpu())
  {
    RT_CHECK(hipSetDevice(dstDev.cuda_device_id()));
  }
  std::vector<double> times;
  for (int i = 0; i < numIters; ++i)
  {
    // Try to get allocation on source
    nvtxRangePush("prefetch to src");
    RT_CHECK(hipMemPrefetchAsync(ptr, count, srcDev.cuda_device_id()));
    RT_CHECK(hipDeviceSynchronize());
    nvtxRangePop();

    // Access from destination and Time
    nvtxRangePush("dst");
    auto start = std::chrono::high_resolution_clock::now();
    if (dstDev.is_cpu())
    {
      cpu_write((int *)ptr, count, stride);
    }
    else
    {
      gpu_write<<<gridDim, blockDim>>>((int *)ptr, count, stride);
      RT_CHECK(hipDeviceSynchronize());
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> txSeconds = end - start;
    nvtxRangePop();
    times.push_back(txSeconds.count());
  }

  const double minTime = *std::min_element(times.begin(), times.end());
  // const double avgTime = std::accumulate(times.begin(), times.end(), 0.0) / times.size();

  printf(",%.2f", count / 1024.0 / 1024.0 / minTime);
  RT_CHECK(hipFree(ptr));
}

int main(int argc, char **argv)
{
  int numIters = 10;
  std::vector<int> numaIds, gpuIds;
  option_as_int(argc, argv, "-n", numIters);
  option_as_int_list(argc, argv, "-c", numaIds);
  option_as_int_list(argc, argv, "-g", gpuIds);

  auto gpus = get_gpus(gpuIds);
  auto cpus = get_cpus(numaIds);

  auto devs = gpus;
  for (auto c : cpus)
  {
    devs.push_back(c);
  }

  const size_t freeMem = free_memory(devs);
  const long pageSize = sysconf(_SC_PAGESIZE);
  auto counts = Sequence::geometric(2048, freeMem, 2) |
                Sequence::geometric(2048 * 1.5, freeMem, 2);

  // print header
  printf("Transfer Size (MB),");
  for (const auto src : devs)
  {
    for (const auto dst : devs)
    {
      if (src != dst && !(src.is_cpu() && dst.is_cpu()))
      {
        printf("%s to %s (coherence),", src.name().c_str(), dst.name().c_str());
      }
    }
  }
  printf("\n");

  for (auto count : counts)
  {
    printf("%f", count / 1024.0 / 1024.0);
    for (const auto src : devs)
    {
      for (const auto dst : devs)
      {
        if (src != dst && !(src.is_cpu() && dst.is_cpu()))
        {
          coherence_bw(dst, src, count, pageSize, numIters);
        }
      }
    }
    printf("\n");
  }

  return 0;
}
