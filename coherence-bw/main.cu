#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <sstream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <numeric>

#include <nvToolsExt.h>

#include <unistd.h>

#include "common/common.hpp"

template <typename data_type>
void cpu_write(data_type *ptr, const size_t count, const size_t stride)
{

  const size_t numElems = count / sizeof(data_type);
  const size_t elemsPerStride = stride / sizeof(data_type);

  for (size_t i = 0; i < numElems; i += elemsPerStride)
  {
    ptr[i] = i * 31ul + 7ul;
  }
}

template <typename data_type, bool NOOP = false>
__global__ void gpu_write(data_type *ptr, const size_t count, const size_t stride)
{
  if (NOOP)
  {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = gx & 31;
  // warp ID
  size_t wx = gx / 32;
  // number of warps in the grid
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;
  // number of strides in N bytes
  const size_t numStrides = count / stride;
  const size_t numData = count / sizeof(data_type);
  // number of data_types in each
  const size_t dataPerStride = stride / sizeof(data_type);

  if (0 == lx)
  {

    for (; wx < numStrides; wx += numWarps)
    {
      const size_t id = wx * dataPerStride;
      if (id < numData)
      {
        ptr[id] = id * 31ul + 7ul;
      }
    }
  }
}

static void prefetch_bw(const int dstDev, const int srcDev, const size_t count, const size_t stride)
{

  if (srcDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(srcDev));
    RT_CHECK(hipFree(0));
  }

  if (dstDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(dstDev));
    RT_CHECK(hipFree(0));
  }

  if (srcDev != hipCpuDeviceId)
  {
    RT_CHECK(hipSetDevice(srcDev));
  }

  // Determine grid dimensions
  dim3 blockDim(256);
  const size_t numStrides = (count + stride - 1) / stride;
  dim3 gridDim((numStrides + blockDim.x - 1) / blockDim.x);

  void *ptr;

  RT_CHECK(hipMallocManaged(&ptr, count));

  std::vector<double> times;
  const size_t numIters = 20;
  for (size_t i = 0; i < numIters; ++i)
  {
    // Try to get allocation on source
    nvtxRangePush("prefetch to src");
    RT_CHECK(hipMemPrefetchAsync(ptr, count, srcDev));
    RT_CHECK(hipDeviceSynchronize());
    nvtxRangePop();

    if (dstDev != hipCpuDeviceId)
    {
      RT_CHECK(hipSetDevice(dstDev));
    }

    // Access from Device and Time
    nvtxRangePush("dst");
    auto start = std::chrono::high_resolution_clock::now();
    if (hipCpuDeviceId == dstDev)
    {
      cpu_write((int *)ptr, count, stride);
    }
    else
    {
      gpu_write<<<gridDim, blockDim>>>((int *)ptr, count, stride);
      RT_CHECK(hipDeviceSynchronize());
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> txSeconds = end - start;
    nvtxRangePop();
    times.push_back(txSeconds.count());
  }

  const double minTime = *std::min_element(times.begin(), times.end());
  const double avgTime = std::accumulate(times.begin(), times.end(), 0.0) / times.size();

  std::cout << "," << count / 1024.0 / 1024.0 / (minTime);
  RT_CHECK(hipFree(ptr));
}

int main(void)
{

  const long pageSize = sysconf(_SC_PAGESIZE);

  int numDevs;
  RT_CHECK(hipGetDeviceCount(&numDevs));

  std::vector<int> devIds;
  for (int dev = 0; dev < numDevs; ++dev)
  {
    devIds.push_back(dev);
  }
  devIds.push_back(hipCpuDeviceId);

  // print header
  std::cout << "Transfer Size (MB),";
  for (const auto src : devIds)
  {
    for (const auto dst : devIds)
    {
      if (src != dst)
      {
        std::cout << src << ":" << dst << ",";
      }
    }
  }
  std::cout << "\n";

  for (size_t count = 2048; count <= 4 * 1024ul * 1024ul * 1024ul; count *= 2)
  {
    std::cout << count / 1024.0 / 1024.0;
    for (const auto src : devIds)
    {
      for (const auto dst : devIds)
      {
        if (src != dst)
        {
          prefetch_bw(dst, src, count, pageSize);
        }
      }
    }
    std::cout << "\n";
  }

  return 0;
}
