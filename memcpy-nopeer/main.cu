#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <numeric>
#include <sstream>
#include <vector>

#include <numa.h>

#include <nvToolsExt.h>

#include <unistd.h>

#include "common/cuda_check.hpp"
#include "common/common.hpp"

static void gpu_gpu_bw(const Device &dst, const Device &src, const size_t count)
{

  assert(src.is_gpu() && dst.is_gpu());

  void *srcPtr, *dstPtr;

  RT_CHECK(hipSetDevice(src.id()));
  RT_CHECK(hipMalloc(&srcPtr, count));
  {
    hipError_t err = hipDeviceDisablePeerAccess(dst.id());
    if (err != hipErrorPeerAccessNotEnabled)
    {
      RT_CHECK(err);
    }
  }
  RT_CHECK(hipSetDevice(dst.id()));
  RT_CHECK(hipMalloc(&dstPtr, count));
  {
    hipError_t err = hipDeviceDisablePeerAccess(src.id());
    if (err != hipErrorPeerAccessNotEnabled)
    {
      RT_CHECK(err);
    }
  }

  std::vector<double> times;
  const size_t numIters = 20;
  for (size_t i = 0; i < numIters; ++i)
  {
    nvtxRangePush("dst");
    auto start = std::chrono::high_resolution_clock::now();
    RT_CHECK(hipMemcpy(dstPtr, srcPtr, count, hipMemcpyDefault));
    RT_CHECK(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> txSeconds = end - start;
    nvtxRangePop();
    times.push_back(txSeconds.count());
  }

  const double minTime = *std::min_element(times.begin(), times.end());
  const double avgTime =
      std::accumulate(times.begin(), times.end(), 0.0) / times.size();

  printf(",%.2f", count / 1024.0 / 1024.0 / minTime);
  RT_CHECK(hipFree(srcPtr));
  RT_CHECK(hipFree(dstPtr));
}

int main(void)
{

  const size_t numNodes = numa_max_node();

  const long pageSize = sysconf(_SC_PAGESIZE);

  std::vector<Device> gpus = get_gpus();

  // print header
  printf("Transfer Size (MB)");
  for (const auto dst : gpus)
  {
    for (const auto src : gpus)
    {
      if (src != dst)
      {
        printf(",%s to %s (no peer)", src.name().c_str(), dst.name().c_str());
      }
    }
  }

  printf("\n");

  auto freeMem = gpu_free_memory(gpus);
  auto counts = Sequence::geometric(2048, freeMem, 2) |
                Sequence::geometric(2048 * 1.5, freeMem, 2);

  for (auto count : counts)
  {
    printf("%f", count / 1024.0 / 1024.0);
    for (const auto dst : gpus)
    {
      for (const auto src : gpus)
      {

        if (src != dst)
        {

          gpu_gpu_bw(dst, src, count);
        }
      }
    }

    printf("\n");
  }

  return 0;
}
