#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <sstream>
#include <cstdlib>
#include <chrono>

#include <nvToolsExt.h>

#include <unistd.h>

#include "common/common.hpp"

size_t cpu_touch(char *c, const size_t e, const size_t n)
{
  for (size_t i = 0; i < n; i += e)
  {
    c[i] = i * 31ul + 7ul;
  }
  return (n / e);
}

template <typename data_type>
__global__ void gpu_touch(data_type *c, const size_t stride, const size_t n, const bool noop = false)
{
  if (noop)
  {
    return;
  }

  // global ID
  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  // lane ID 0-31
  const size_t lx = (blockDim.x * blockIdx.x + threadIdx.x) & 31;
  // warp ID
  size_t wx = gx / 32;
  // number of warps in the grid
  const size_t numWarps = (gridDim.x * blockDim.x + 32 - 1) / 32;
  // number of strides in N bytes
  const size_t numStrides = (n + stride - 1) / stride;
  // number of data_types in each
  const size_t elemsPerStride = stride / sizeof(data_type);

  if (0 == lx)
  {

    for (; wx < numStrides; wx += numWarps)
    {
      const size_t id = wx * elemsPerStride;
      if (id < numStrides)
      {
        c[id] = id * 31ul + 7ul;
      }
    }
  }
}

int main(void)
{

  const long pageSize = sysconf(_SC_PAGESIZE);
  std::stringstream buffer;

  RT_CHECK(hipFree(0));

  size_t memTotal, memAvail;
  RT_CHECK(hipMemGetInfo(&memAvail, &memTotal));

  char *cm;

  RT_CHECK(hipMallocManaged(&cm, pageSize * 32));
  RT_CHECK(hipDeviceSynchronize());

  const int numIters = 80;

  for (size_t n = 4; n <= pageSize * 32; n *= 2)
  {
    buffer.str("");
    buffer << n;
    nvtxRangePush(buffer.str().c_str());

    RT_CHECK(hipMallocManaged(&cm, n));
    RT_CHECK(hipDeviceSynchronize());

    const size_t stride = 4096;

    // create enough warps to cover all the strides
    const size_t numThreads = 32 * ((n + stride - 1) / stride);
    dim3 dimBlock(128);
    dim3 dimGrid((numThreads + 128 - 1) / 128);
    // std::cout << numThreads << " " << dimGrid.x << "\n";

    // Loop with work
    nvtxRangePush("work");
    auto start = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < numIters; ++i)
    {

      nvtxRangePush("cpu");
      cpu_touch(cm, stride, n);
      RT_CHECK(hipDeviceSynchronize());
      nvtxRangePop();
      nvtxRangePush("gpu");
      gpu_touch<<<dimGrid, dimBlock>>>(cm, stride, n);
      RT_CHECK(hipDeviceSynchronize());
      nvtxRangePop();
    }
    auto end = std::chrono::high_resolution_clock::now();
    nvtxRangePop();
    std::chrono::duration<double> workSeconds = end - start;

    // empty loop
    nvtxRangePush("noop");
    start = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < numIters; ++i)
    {
      nvtxRangePush("cpu");
      RT_CHECK(hipDeviceSynchronize());
      nvtxRangePop();
      nvtxRangePush("gpu");
      gpu_touch<<<dimGrid, dimBlock>>>(cm, stride, n, true /*no-op*/);
      RT_CHECK(hipDeviceSynchronize());
      nvtxRangePop();
    }
    end = std::chrono::high_resolution_clock::now();
    nvtxRangePop();
    std::chrono::duration<double> emptySeconds = end - start;

    std::cout << "n=" << n << ": " << (workSeconds.count() - emptySeconds.count()) / numIters << " s/iter (" << numIters << ")\n";

    RT_CHECK(hipFree(cm));
    nvtxRangePop();
  }

  return 0;
}
