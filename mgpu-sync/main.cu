#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sstream>

#include <nvToolsExt.h>
#include <hip/hip_cooperative_groups.h>

#include <unistd.h>

#include "common/common.hpp"

template <size_t NUM_ITERS>
__global__ void gpu_sync(long long *clocks, const int dev)
{

  using namespace cooperative_groups;

  const size_t gx = blockDim.x * blockIdx.x + threadIdx.x;

  multi_grid_group mg = this_multi_grid();

  long long start = clock64();
#pragma unroll(NUM_ITERS)
  for (int iter = 0; iter < NUM_ITERS; ++iter)
  {
    mg.sync();
  }
  long long end = clock64();

  clocks[gx] = (end - start) / NUM_ITERS;
}

std::vector<void *> box(long long *clocks, const int dev)
{

#define BOX(v, T)                            \
  {                                          \
    auto box = new T;                        \
    *box = v;                                \
    ret.push_back(static_cast<void *>(box)); \
  }

  std::vector<void *> ret;

  BOX(clocks, long long *);
  BOX(dev, int);

#undef BOX

  return ret;
}

int main(void)
{

  std::vector<int> devices = {0, 1};

  // create streams
  std::vector<hipStream_t> streams(2);
  for (size_t i = 0; i < streams.size(); ++i)
  {
    RT_CHECK(hipSetDevice(i));
    RT_CHECK(hipStreamCreate(&streams[i]));
  }

  // determine kernel parameters
  dim3 gridDim(1);
  dim3 blockDim(32);
  const size_t numThreads = gridDim.x * blockDim.x;

  // allocate clock arrays
  std::vector<long long *> devClocks(2);
  std::vector<long long *> hostClocks(2);
  for (size_t dev = 0; dev < devClocks.size(); ++dev)
  {
    RT_CHECK(hipSetDevice(dev));
    RT_CHECK(hipMalloc(&devClocks[dev], numThreads * sizeof(long long)));
    hostClocks[dev] = new long long[numThreads];
  }

  // create argument lists
  std::vector<std::vector<void *>> kernelArgsList(2);
  for (size_t dev = 0; dev < kernelArgsList.size(); ++dev)
  {
    kernelArgsList[dev] = box(devClocks[dev], dev);
  }

  // create launch parameters lists
  std::vector<hipLaunchParams> paramsList(2);

  for (size_t i = 0; i < paramsList.size(); ++i)
  {
    auto &params = paramsList[i];

    params.func = (void *)gpu_sync<1>;
    params.gridDim = gridDim;
    params.blockDim = blockDim;
    params.args = &(kernelArgsList[i][0]);
    params.sharedMem = 0;
    params.stream = streams[i];
  }

  RT_CHECK(hipLaunchCooperativeKernelMultiDevice(&paramsList[0], 2));

  // copy clocks back to host
  for (size_t dev = 0; dev < devClocks.size(); ++dev)
  {
    RT_CHECK(hipMemcpy(hostClocks[dev], devClocks[dev], numThreads * sizeof(long long), hipMemcpyDefault));
  }

  // Print some host clocks:
  for (size_t dev = 0; dev < devClocks.size(); ++dev)
  {
    std::cout << "dev: " << dev << std::endl;
    for (size_t i = 0; i < numThreads; ++i)
    {
      std::cerr << hostClocks[dev][i] << " ";
    }
    std::cout << std::endl;
  }

  nvtxRangePush("cleanup");
  for (auto &stream : streams)
  {
    RT_CHECK(hipStreamDestroy(stream));
  }

  // for (auto &kernelArgs : kernelArgsList)
  // {
  //   for (auto &arg : kernelArgs)
  //   {
  //     delete arg;
  //     arg = nullptr;
  //   }
  // }

  nvtxRangePop();

  return 0;
}
